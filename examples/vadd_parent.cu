
#include <hip/hip_runtime.h>
extern "C" {

__device__ float device_add(float a, float b);

__global__ void kernel_vadd(const float *a, const float *b, float *c)
{
    int i = blockIdx.x *blockDim.x + threadIdx.x;
    c[i] = device_add(a[i], b[i]);
}

}
